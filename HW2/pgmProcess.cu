#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "pgmProcess.h"
/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
    int distanceX = p2[1] - p1[1];
    int distanceY = p2[0] - p1[0];
    return sqrt((float)(distanceX * distanceX) + (float)(distanceY * distanceY));
    return 0.0;
}

int  pgmDrawEdge(int *pixels, int numRows, int numCols, int edgeWidth, char **header) {
    int* dPixels;
    int blockSize = 64;
    int gridSize = ceil(((double)numRows * (double)numCols) / (double) blockSize);
    int arraySizeInBytes = sizeof(int) * numRows * numCols;

    // allocate device memory for the array
    hipMalloc(&dPixels, arraySizeInBytes);

    // copy the cpu memory to the gpu
    hipMemcpy(dPixels, pixels, arraySizeInBytes, hipMemcpyHostToDevice);

    // run the kernel
    gpuDrawEdge<<<gridSize, blockSize>>>(dPixels, numRows, numCols, edgeWidth, numRows * numCols);

    // copy the results back to the host array
    hipMemcpy(pixels, dPixels, arraySizeInBytes, hipMemcpyDeviceToHost);

    // release the device array
    hipFree(dPixels);
    return 0;
}

int pgmDrawCircle(int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header) {

    int* dPixels;
    int blockSize = 64;
    int gridSize = ceil(((double)numRows * (double)numCols) / (double) blockSize);
    int arraySizeInBytes = sizeof(int) * numRows * numCols;

    // allocate device memory for the array
    hipMalloc(&dPixels, arraySizeInBytes);

    // copy the cpu memory to the gpu
    hipMemcpy(dPixels, pixels, arraySizeInBytes, hipMemcpyHostToDevice);

    // run the kernel
    gpuDrawCircle<<<gridSize, blockSize>>>(dPixels, numRows, numCols, centerRow, centerCol, radius, numRows * numCols);

    // copy the results back to the host array
    hipMemcpy(pixels, dPixels, arraySizeInBytes, hipMemcpyDeviceToHost);

    // release the device array
    hipFree(dPixels);

    return 0;
}

int pgmDrawLine(int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col){
    int* dPixels;
    int blockSize = 64;
    int arraySizeInBytes = sizeof(int) * numRows * numCols;

    int pa[] = {0,0};
    int pb[] = {0,0};
    int vertical = 0;
    float slope = 0.0;
    if(p1col < p2col){
        pa[0] = p1row;
        pa[1] = p1col;
        pb[0] = p2row;
        pb[1] = p2col;
    }
    else{
        pa[0] = p2row;
        pa[1] = p2col;
        pb[0] = p1row;
        pb[1] = p1col;
    }

    if(pa[1] == pb[1])
        vertical = 1;
    else
        slope = (pb[0] - pa[0]) / (pb[1] - pa[1]);

    int startCol = pa[1];
    int noSamples = (vertical) ? ((pa[0] < pb[0]) ? pb[0] - pa[0] : pa[0] - pb[0]) : pb[1] - pa[0];

    int gridSize = ceil(((double)noSamples) / (double) blockSize);

    // allocate device memory for the array
    hipMalloc(&dPixels, arraySizeInBytes);

    // copy the cpu memory to the gpu
    hipMemcpy(dPixels, pixels, arraySizeInBytes, hipMemcpyHostToDevice);

    // run the kernel
    gpuDrawLine<<<gridSize, blockSize>>>(dPixels, noRows, noCol, pa[0], pa[1], vertical, slope, noSamples);

    // copy the results back to the host array
    hipMemcpy(pixels, dPixels, arraySizeInBytes, hipMemcpyDeviceToHost);

    // release the device array
    hipFree(dPixels);

    return 0;
}

__global__ void gpuDrawEdge(int *pixels, int numRows, int numCols, int edgeWidth, int n) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int thisRow  = threadId / numCols;
    int thisCol  = threadId % numCols;

    if(thisRow <= edgeWidth ||
       thisRow >= numRows - edgeWidth ||
       thisCol <= edgeWidth ||
       thisCol >= numCols - edgeWidth){
        pixels[threadId] = 0;
    }
}

__global__ void gpuDrawCircle(int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, int n) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int thisRow  = threadId / numCols;
    int thisCol  = threadId % numCols;

    int p1[] = {thisRow, thisCol};
    int p2[] = {centerRow, centerCol};

    if(distance(p1, p2) <= radius)
        pixels[threadId] = 0;
}

__global__ void gpuDrawLine(int *pixels, int noRows, int noCols, int startRow, int startCol, int vertical, float slope, int noSamples);
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int position = startCol + threadId;
    int thisRow, thisCol;
    if(vertical){
        thisRow = startRow + position;
        thisCol = startCol;
    }
    else{
        thisCol = startCol + position;
        thisRow = startRow + (slope * startRow);
    }
    int index = noCols * thisRow + thisCol;
    pixels[index] = 0;
}



#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "pgmProcess.h"
/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
    //int distanceX = p2[1] - p1[1];
    //int distanceY = p2[0] - p1[0];
    //return sqrt((float)(distanceX * distanceX) + (float)(distanceY * distanceY));
    return 0.0;
}

int  pgmDrawEdge(int *pixels, int numRows, int numCols, int edgeWidth, char **header) {
    int* dPixels;
    int blockSize = 512;
    int gridSize = ceil(((double)numRows * (double)numCols) / (double) blockSize);
    int arraySizeInBytes = sizeof(int) * numRows * numCols;

    // allocate device memory for the array
    hipMalloc(&dPixels, arraySizeInBytes);

    // zero the memory in cuda
    //hipMemset(d_array, 0, arraySizeInBytes);

    // copy the cpu memory to the gpu
    hipMemcpy(dPixels, pixels, arraySizeInBytes, hipMemcpyHostToDevice);

    // run the kernel
    gpuDrawEdge<<<gridSize, blockSize>>>(dPixels, numRows, numCols, edgeWidth);

    // copy the results back to the host array
    hipMemcpy(pixels, dPixels, arraySizeInBytes, hipMemcpyDeviceToHost);

    // release the device array
    hipFree(dPixels);
    return 0;
}

int pgmDrawCircle(int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header) {
    return 0;
}

int pgmDrawLine(int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col){
    return 0;
}

__host__ __device__ void  gpuDrawEdge(int *pixels, int numRows, int numCols, int edgeWidth) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int thisRow  = threadId / numCols;
    int thisCol  = threadId % numCols;

    if(thisRow <= edgeWidth ||
       thisRow >= numCols - edgeWidth ||
       thisCol <= edgeWidth ||
       thisCol >= numRows - edgeWidth){
        pixels[threadId] = 0;
    }
}

__host__ __device__ void gpuDrawCircle(int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int thisRow  = threadId / numCols;
    int thisCol  = threadId % numCols;


}

__host__ __device__ void gpuDrawLine(int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col) {

}


